/*

nvcc -o hpc4B hpc4B.cu
./hpc4B

Enter the size of the square matrix: 2
Enter elements of matrix A:
1 2
3 4
Enter elements of matrix B:
5 6
7 8
Result of matrix multiplication:
19 22
43 50

*/

#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

// CUDA kernel function for matrix multiplication
__global__ void matrixMul(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread is within bounds of the matrix
    if (row < n && col < n) {
        int sum = 0;
        // Perform the dot product of row and column
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int N;
    cout << "Enter the size of the square matrix: "; 
    cin >> N;

    // Allocate memory for matrices A, B, and C on the host (CPU)
    int *h_A = new int[N * N], *h_B = new int[N * N], *h_C = new int[N * N];

    cout << "Enter elements of matrix A:\n";
    for (int i = 0; i < N * N; i++) {
        cin >> h_A[i];  // Input elements for matrix A
    }

    cout << "Enter elements of matrix B:\n";
    for (int i = 0; i < N * N; i++) {
        cin >> h_B[i];  // Input elements for matrix B
    }

    // Allocate memory for matrices A, B, and C on the device (GPU)
    int *d_A, *d_B, *d_C;
    int size = N * N * sizeof(int);

    hipError_t err = hipMalloc(&d_A, size);  // Allocate device memory for A
    if (err != hipSuccess) {
        cerr << "CUDA malloc failed for A: " << hipGetErrorString(err) << endl;
        return -1;
    }

    err = hipMalloc(&d_B, size);  // Allocate device memory for B
    if (err != hipSuccess) {
        cerr << "CUDA malloc failed for B: " << hipGetErrorString(err) << endl;
        return -1;
    }

    err = hipMalloc(&d_C, size);  // Allocate device memory for C
    if (err != hipSuccess) {
        cerr << "CUDA malloc failed for C: " << hipGetErrorString(err) << endl;
        return -1;
    }

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Set the number of threads per block (16x16 block)
    dim3 threadsPerBlock(16, 16);
    // Calculate the number of blocks per grid
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);

    // Launch the kernel to perform matrix multiplication
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << endl;
        return -1;
    }

    // Copy the result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Output the result of matrix multiplication
    cout << "Result of matrix multiplication:\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cout << h_C[i * N + j] << " ";  // Print each element of the result matrix
        }
        cout << "\n";
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
